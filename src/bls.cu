#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <climits>
#include <cmath>
#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <vector>

using namespace std;

struct BlsResult {
  double period;
  double d_value;
};

__global__ void weight_sum_kernel(const double *flux_err, double *weight,
                                  int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n) {
    atomicAdd(weight, pow(flux_err[idx], -2));
  }
}

__global__ void compute_weight(const double *flux_err, double *weight,
                               double sum_w, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n) {
    weight[idx] = sum_w * pow(flux_err[idx], -2);
  }
}

__global__ void bls_kernel(const double *time, const double *flux,
                           const double *weight, int n, BlsResult *result) {
  int idx1 = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx1 >= n)
    return;

  for (int idx2 = idx1 + 1; idx2 < n; idx2++) {
    double r = 0, s = 0, aux = 0;
    for (int i = idx1; i <= idx2; i++) {
      r += weight[i];
      s += weight[i] * flux[i];
    }
    for (int i = 0; i < n; i++) {
      aux += weight[i] * pow(flux[i], 2);
    }
    double d = aux - pow(s, 2) / (r * (1 - r));
    double period = time[idx2] - time[idx1];
    if (d < result->d_value) {
      result->d_value = d;
      result->period = period;
    }
  }
}

void readCSV(const string &filename, vector<double> &time, vector<double> &flux,
             vector<double> &flux_err) {
  ifstream file(filename);
  string line;

  if (!file.is_open()) {
    cerr << "Error opening file: " << filename << endl;
    return;
  }

  // Read the header line
  if (getline(file, line)) {
    // Do nothing with the header line
  }

  while (getline(file, line)) {
    stringstream lineStream(line);
    string cell;
    vector<string> row;

    while (getline(lineStream, cell, ',')) {
      row.push_back(cell);
    }

    if (row.size() >= 3) { // Ensure there are at least 3 columns
      time.push_back(stod(row[0]));
      flux.push_back(stod(row[1]));
      flux_err.push_back(stod(row[2]));
    }
  }

  file.close();
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cout << "Usage: " << argv[0] << " <filename>" << endl;
    return 1;
  }

  string filename = argv[1];
  vector<double> time, flux, flux_err;

  readCSV(filename, time, flux, flux_err);

  int n = flux.size();

  thrust::host_vector<double> h_time = time;
  thrust::host_vector<double> h_flux = flux;
  thrust::host_vector<double> h_flux_err = flux_err;

  thrust::device_vector<double> d_time = h_time;
  thrust::device_vector<double> d_flux = h_flux;
  thrust::device_vector<double> d_flux_err = h_flux_err;
  thrust::device_vector<double> d_weight(n);

  double h_weight_sum = 0;
  double *d_weight_sum;

  hipMalloc((void **)&d_weight_sum, sizeof(double));
  hipMemcpy(d_weight_sum, &h_weight_sum, sizeof(double),
             hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  if (n % blockSize != 0) {
    numBlocks++;
  }

  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(blockSize, 1, 1);

  auto start = chrono::high_resolution_clock::now();

  weight_sum_kernel<<<dimGrid, dimBlock>>>(
      thrust::raw_pointer_cast(d_flux_err.data()), d_weight_sum, n);
  hipMemcpy(&h_weight_sum, d_weight_sum, sizeof(double),
             hipMemcpyDeviceToHost);

  h_weight_sum = pow(h_weight_sum, -1);
  compute_weight<<<dimGrid, dimBlock>>>(
      thrust::raw_pointer_cast(d_flux_err.data()),
      thrust::raw_pointer_cast(d_weight.data()), h_weight_sum, n);

  BlsResult h_result;
  h_result.d_value = LONG_MAX;
  BlsResult *d_result;

  hipMalloc((void **)&d_result, sizeof(BlsResult));
  hipMemcpy(d_result, &h_result, sizeof(BlsResult), hipMemcpyHostToDevice);

  bls_kernel<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(d_time.data()),
                                    thrust::raw_pointer_cast(d_flux.data()),
                                    thrust::raw_pointer_cast(d_weight.data()),
                                    n, d_result);
  hipMemcpy(&h_result, d_result, sizeof(BlsResult), hipMemcpyDeviceToHost);

  auto end = chrono::high_resolution_clock::now();
  auto duration = chrono::duration_cast<chrono::seconds>(end - start);

  cout << "Period: " << h_result.period << endl;
  cout << "D Value: " << h_result.d_value << endl;
  cout << "Time: " << duration.count() << " s" << endl;

  hipFree(d_weight_sum);
  hipFree(d_result);

  return 0;
}
