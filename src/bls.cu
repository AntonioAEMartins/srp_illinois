#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <climits>
#include <cmath>
#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <vector>

using namespace std;

struct BlsResult {
    double period;
    double d_value;
};

struct WeightSum {
    __host__ __device__ double operator()(const double &x) const {
        return pow(x, -2);
    }
};

struct WeightCalculation {
    double sum_w;
    WeightCalculation(double _sum_w) : sum_w(_sum_w) {}

    __host__ __device__ double operator()(const double &x) const {
        return sum_w * pow(x, -2);
    }
};

struct RValue {
    double *weight;
    int i1, i2;
    RValue(double *_weight, int _i1, int _i2) : weight(_weight), i1(_i1), i2(_i2) {}

    __host__ __device__ double operator()(const int &x) const {
        double r = 0;
        for (int i = i1; i <= i2; i++) {
            r += weight[i];
        }
        return r;
    }
};

struct SValue {
    double *weight;
    double *flux;
    int i1, i2;
    SValue(double *_weight, double *_flux, int _i1, int _i2) : weight(_weight), flux(_flux), i1(_i1), i2(_i2) {}

    __host__ __device__ double operator()(const int &x) const {
        double s = 0;
        for (int i = i1; i <= i2; i++) {
            s += (weight[i] * flux[i]);
        }
        return s;
    }
};

struct DValue {
    double *weight;
    double *flux;
    double r, s;
    int size;
    DValue(double *_weight, double *_flux, double _r, double _s, int _size) : weight(_weight), flux(_flux), r(_r), s(_s), size(_size) {}

    __host__ __device__ double operator()(const int &x) const {
        double aux = 0;
        for (int i = 0; i < size; i++) {
            aux += (weight[i] * pow(flux[i], 2));
        }
        return aux - pow(s, 2) / (r * (1 - r));
    }
};

BlsResult my_bls(thrust::device_vector<double> &time, thrust::device_vector<double> &flux, thrust::device_vector<double> &flux_err) {
    BlsResult result;
    result.d_value = DBL_MAX;

    double sum_w = thrust::transform_reduce(flux_err.begin(), flux_err.end(), WeightSum(), 0.0, thrust::plus<double>());

    thrust::device_vector<double> weight(flux.size());
    thrust::transform(flux_err.begin(), flux_err.end(), weight.begin(), WeightCalculation(sum_w));

    for (int i1 = 0; i1 < flux.size(); i1++) {
        for (int i2 = i1 + 1; i2 < flux.size(); i2++) {
            double r_ = thrust::transform_reduce(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(flux.size()), RValue(thrust::raw_pointer_cast(weight.data()), i1, i2), 0.0, thrust::plus<double>());
            double s_ = thrust::transform_reduce(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(flux.size()), SValue(thrust::raw_pointer_cast(weight.data()), thrust::raw_pointer_cast(flux.data()), i1, i2), 0.0, thrust::plus<double>());
            double d_ = thrust::transform_reduce(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(flux.size()), DValue(thrust::raw_pointer_cast(weight.data()), thrust::raw_pointer_cast(flux.data()), r_, s_, flux.size()), 0.0, thrust::plus<double>());

            double period = (time[i2] - time[i1]);

            if (d_ < result.d_value) {
                result.d_value = d_;
                result.period = period;
            }

            cout << "i1: " << i1 << " i2: " << i2 << " period: " << period << " d: " << d_ << endl;
        }
    }

    return result;
}

void readCSV(const string &filename, thrust::host_vector<double> &time, thrust::host_vector<double> &flux, thrust::host_vector<double> &flux_err) {
    ifstream file(filename);
    string line;

    if (!file.is_open()) {
        cerr << "Error opening file: " << filename << endl;
        return;
    }

    if (getline(file, line)) {
        // Do nothing with the header line
    }

    while (getline(file, line)) {
        stringstream lineStream(line);
        string cell;
        vector<string> row;

        while (getline(lineStream, cell, ',')) {
            row.push_back(cell);
        }

        if (row.size() >= 3) {
            time.push_back(stod(row[0]));
            flux.push_back(stod(row[1]));
            flux_err.push_back(stod(row[2]));
        }
    }

    file.close();
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        cout << "Usage: " << argv[0] << " <filename>" << endl;
        return 1;
    }

    string filename = argv[1];
    thrust::host_vector<double> h_time, h_flux, h_flux_err;

    readCSV(filename, h_time, h_flux, h_flux_err);

    thrust::device_vector<double> d_time = h_time;
    thrust::device_vector<double> d_flux = h_flux;
    thrust::device_vector<double> d_flux_err = h_flux_err;

    auto start = chrono::high_resolution_clock::now();

    BlsResult result = my_bls(d_time, d_flux, d_flux_err);

    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::seconds>(end - start);

    cout << "Period: " << result.period << endl;
    cout << "D Value: " << result.d_value << endl;
    cout << "Time: " << duration.count() << " s" << endl;

    return 0;
}
